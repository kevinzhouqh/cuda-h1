#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <stdbool.h>
#include <sys/time.h>
double getTimeStamp() {
	struct timeval tv;
	gettimeofday( &tv, NULL );
	return (double) tv.tv_usec/1000000 + tv.tv_sec;
}

void h_addmat(float*A, float*B, float*C, int nx, int ny)
{
	int total = nx*ny;
	int count = 0;
	int i;
	for(i=0; i<total; i++)
	{
		C[count] = A[count] + B[count];
		count++;
	}
	return;
}

__global__ void f_addmat( float*A, float*B, float*C, int nx, int ny) {
	int ix = threadIdx.x + blockIdx.x*blockDim.x ;
	int iy = threadIdx.y + blockIdx.y*blockDim.y ;
	int idx = ix*ny + iy;
	if ( (ix<nx) && (iy<ny))
		C[idx] = A[idx] + B[idx];
}

int main( int argc, char *argv[] ) {
	if (argc != 3)
	{
		printf("Error: wrong number\n");
		exit(0);
	}

	int nx = atoi ( argv[1] );
	int ny = atoi (argv[2] );
	int noElems = nx*ny;
	int bytes = noElems * sizeof(float);
	int i,j, count;
	count = 0;
	float *h_A = (float *) malloc ( bytes );
	float* h_B = (float *) malloc ( bytes );
	float *h_hC = (float *) malloc ( bytes );
	float *h_dC = (float *) malloc ( bytes );
	
	for (i=0; i<nx; i++)
		for (j=0; j<ny; j++)
		{
			h_A[count] = (float)(i+j)/3.0;
			count++;
		}
	count = 0;
	for (i=0; i<nx; i++)
		for (j=0; j<ny; j++)
		{
			h_B[count]= (float)3.14*(i+j);
			count++;
		}
        float *d_A, *d_B, *d_C ;
	hipMalloc( (void **) &d_A, bytes);
	hipMalloc( (void **) &d_B, bytes);
	hipMalloc( (void **) &d_C, bytes);
	double timeStampA = getTimeStamp();
	hipMemcpy( d_A, h_A, bytes, hipMemcpyHostToDevice);
	hipMemcpy( d_B, h_B, bytes, hipMemcpyHostToDevice);
	double timeStampB = getTimeStamp();
	dim3 block(128, 8);
	dim3 grid((nx + block.x-1)/block.x, (ny+block.y-1)/block.y);
	f_addmat<<<grid, block>>>( d_A, d_B, d_C, nx, ny);
	hipDeviceSynchronize();
	double timeStampC = getTimeStamp();
	hipMemcpy(h_dC, d_C, bytes, hipMemcpyDeviceToHost );
	double timeStampD = getTimeStamp();
	hipFree( d_A ); hipFree( d_B); hipFree( d_C);
	hipDeviceReset();
	h_addmat(h_A, h_B, h_hC, nx, ny);
	count = 0;
	bool s = true;
	for(i=0; i<noElems; i++)
	{
		if( h_hC[i] != h_dC[i] )
		{
			s = false;
			printf("%d \n", i);
			break;
		}
	}
	if(s)
	{
		printf("total time is %f, CPU GPU transfer time is %f, kernel time is %f, GPU CPU transfer time is %f\n ", timeStampD-timeStampA, timeStampB - timeStampA, timeStampC- timeStampB, timeStampD - timeStampC);
		exit(0);
	}
	printf("finished");
	return 0;

}
